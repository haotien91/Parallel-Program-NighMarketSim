#include "main.cuh"

int main(int argc, char **argv)
{
    char *input_filename = argv[1];
    char *output_filename = argv[2];

    // Initialize objects
    street *Playground = new street;

    // Handle Input
    Playground->Load_map(input_filename);

    Playground->Output_size(output_filename);

    // FOR loop RUN simulator
    dim3 blk(32, 32);
    dim3 grid(MAP_SIZE / 32, MAP_SIZE / 32);

    set<<<grid, blk>>>(Playground->Dstreetmap, Playground->Dscaled_map);

    // test write in

    for (int i = 0; i < PHASES; i++)
    {
        printf("In phase %d\n", i);
        decide<<<grid, blk>>>(Playground->Dscaled_map);
        // test_write<<<grid, blk>>>(Playground->Dstreetmap, Playground->Dscaled_map, Playground->DOutputmap);// print Dscalemap

        run<<<grid, blk>>>(Playground->Dscaled_map);
        // test_write<<<grid, blk>>>(Playground->Dstreetmap, Playground->Dscaled_map, Playground->DOutputmap);// print Dscalemap

        check<<<grid, blk>>>(Playground->Dscaled_map, Playground->DOutputmap);
        // test_write<<<grid, blk>>>(Playground->Dstreetmap, Playground->Dscaled_map, Playground->DOutputmap); // print Dscalemap

        // For : when run finish a phase , trigger event

        Playground->Output_map(output_filename);
        /*
                for(int i = 0 ; i < MAP_SIZE ; i++)
                {
                    for(int j = 0 ; j < MAP_SIZE ; j++)
                    {
                        printf("%d ", (Playground->Outputmap)[C(j,i,MAP_SIZE)]);
                    }
                    printf("\n");
                }
        */
    }
    //  free(Playground->Outputmap);

    // free memory
    printf("finished start deleting object \n");

    // std::cout << sizeof(Playground->streetmap) << std::endl;
    // std::cout << sizeof(Playground->Outputmap) << std::endl;

    // printf("Outputfile is %d\n", Playground->Outputmap[6 * 64 + 16]);

    // free(Playground->streetmap);
    // printf("streetmap\n");
    // free(Playground->Outputmap);
    // printf("Output\n");

    delete Playground;
    printf("delete success\n");

    return;
}
