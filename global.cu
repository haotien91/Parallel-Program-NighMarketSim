#include "hip/hip_runtime.h"
#include "global.cuh"

__device__ int preference::set_weights(int * Dx_bounds, int * Dy_bounds,pos position)
{
    // // find left, right bound 
    int inp_size = MAP_SIZE/SCALE_SIZE;
    int left,right,up,down;
    int scale_y = position.y / SCALE_SIZE;
    for(int i = 0 ; i < inp_size ; i++)
    {
        
        if (position.x <= Dx_bounds[C(i,scale_y,inp_size)] && Dx_bounds[C(i,scale_y,inp_size)] != -1)
        {
            right = Dx_bounds[C(i,scale_y,inp_size)];
            if(i ==0 )left = -100000000;
            else left = Dx_bounds[C(i-1,scale_y,inp_size)];


            break;
        }
        if(Dx_bounds[C(i,scale_y,inp_size)]  == -1 )
        {
            right = 100000000;
            if(i ==0)left = -100000000;
            else left = Dx_bounds[C(i-1,scale_y,inp_size)];
            //set to map bound
            break;
        }
    }
    

    int scale_x = position.x /  SCALE_SIZE;
    for(int i = 0 ; i < inp_size ; i++)
    {
        if (position.y <= Dy_bounds[C(i,scale_x,inp_size)] && Dy_bounds[C(i,scale_x,inp_size)] != -1)
        {
            down = Dy_bounds[C(i,scale_x,inp_size)];
            if(i ==0)up = -100000000;
            else up = Dy_bounds[C(i-1,scale_x,inp_size)];

            break;
        }
        if(Dy_bounds[C(i,scale_x,inp_size)] == -1 )
        {
            down = 100000000;
            if(i ==0)up = -100000000;
            else up = Dy_bounds[C(i-1,scale_x,inp_size)];
            //set to map bound
            break;
        }
    }

    int vecs[] = {position.y - up,down - position.y,position.x - left,right - position.x};
    int opp[] = {DOWN,UP,RIGHT,LEFT};
    vecs[BACK] = 0 ;
    vecs[opp[heading]]= 0;
    

    // get max vecs 
    int  max_val = 0 ,max_ind;
    for(int i = 0 ; i< 4; i++)
    {
        if(max_val <= vecs[i])
        {
            max_val = vecs[i];
            max_ind = i;
        }
    }
    
    float turn_prob = 0 ;
    // if diff , get turn prob 
    if(max_ind != this->heading)
    {
     //   printf("%d %d %d %d\n",position.x, position.y ,vecs[max_ind],vecs[heading]);
     //   turn_prob = ( 10 + ( 10 > vecs[heading]) ? (10 - vecs[heading])*10 : 0  );
      turn_prob = ( 15 + ( MAP_SIZE/2 > vecs[heading]) ? (MAP_SIZE/2- vecs[heading])*10 : 0  );
    }
    hiprandState state;
    hiprand_init(clock64(), C(position.x,position.y,MAP_SIZE) , 0, &state);
    float myrandf = hiprand_uniform(&state);

    myrandf *= (100);
 //   printf("%d %d %d %d %d %d\n",position.x , position.y , up , down ,left, right);
    if(myrandf < turn_prob)
    {
        //turn 
        return max_ind;
    }
    else return heading;
}
__device__ int preference::choose(int * Dx_bounds, int * Dy_bounds)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    // Define weighted probabilities for each direction

    int new_heading = set_weights(Dx_bounds,Dy_bounds,position);
    this->heading = new_heading;
    int *  weights = this->heading_weights[new_heading];

    

    // Calculate total weight
    int totalWeight = 0;
    for (int i = 0 ; i< 4; i++)
    {
        totalWeight += weights[i];
    }

    // Generate a random number between 1 and total weight
    //  std::random_device rd;
    //   std::mt19937 gen(rd());
    //   std::uniform_int_distribution<> dis(1, totalWeight);

    // assume have already set up hiprand and generated state for each thread...
    // assume ranges vary by thread index
    hiprandState state;
    hiprand_init(clock64(), C(position.x,position.y,MAP_SIZE) , 0, &state);
    float myrandf = hiprand_uniform(&state);
    myrandf *= (100);
    myrandf += UP;

    int randomNum = (int)truncf(myrandf);
    ; // NEED TO CHANGE

    // Choose a direction based on weighted probabilities
    if (randomNum <= weights[0])
    {
        return UP; // Up
    }
    else if (randomNum <= weights[0] + weights[1])
    {
        return DOWN; // Down
    }
    else if (randomNum <= weights[0] + weights[1] + weights[2])
    {
        return LEFT; // Left
    }
    else
    {
        return RIGHT; // Right
    }
}
