#include "street.cuh"

void 
street::Load_map(char *infile)
{
    FILE *file = fopen(infile, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE; 



    int * tmp = (int*)malloc(inp_size *inp_size);

    for (int i = 0; i < inp_size; i++)
    {
        fread(&tmp, sizeof(int), inp_size * inp_size, file);
    }

    // throw to gpu 

    hipMalloc((void**)&Dstreetmap, inp_size *inp_size * sizeof(int));
    hipMemcpy(Dstreetmap, tmp , inp_size *inp_size * sizeof(int) , hipMemcpyHostToDevice);
}

void 
street::Output_map(char *outfile)
{
    return ;
   
}