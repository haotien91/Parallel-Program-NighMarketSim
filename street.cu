#include "street.cuh"

void street::Load_map(char *infilename)
{
    FILE *file = fopen(infilename, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    this->streetmap = (int *)malloc(inp_size * inp_size *sizeof(int));
    this->Outputmap = (int *)malloc(MAP_SIZE * MAP_SIZE *sizeof(int));

    fread(this->streetmap, sizeof(int), inp_size * inp_size, file);

    for (int i = 0; i < inp_size; i++)
    {
        for (int j = 0; j < inp_size; j++)
        {
            printf("%d ", this->streetmap[C(j, i, inp_size)]);
        }
        printf("\n");
    }

    // throw to gpu

    hipMalloc((void **)&this->Dstreetmap, inp_size * inp_size * sizeof(int));
    hipMemcpy(this->Dstreetmap, this->streetmap, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&this->Dscaled_map, MAP_SIZE * MAP_SIZE * sizeof(map));
    hipMalloc((void **)&this->DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(int));

    printf("loaded input  map , inp size %d \n", inp_size);
}

void street::Output_map(char *outfilename)
{

    hipMemcpy(this->Outputmap, this->DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // if a phase is ended. output current map.
    FILE *outfile = fopen(outfilename, "ab");

    fwrite(this->Outputmap, sizeof(int), MAP_SIZE * MAP_SIZE, outfile);

    fclose(outfile);

    return;
}
