#include "street.cuh"

void street::Load_map(char *infilename)
{
    FILE *file = fopen(infilename, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    streetmap = (int *)malloc(inp_size * inp_size);
    Outputmap = (int *)malloc(MAP_SIZE * MAP_SIZE);

   
    fread(streetmap, sizeof(int), inp_size * inp_size, file);

    
    for(int i =0 ; i< inp_size ;i++)
    {
        for(int j = 0 ; j< inp_size ; j++)
        {
            printf("%d ",streetmap[C(j,i,inp_size)]);
 
        }
        printf("\n");
    }

    // throw to gpu

    hipMalloc((void **)&Dstreetmap, inp_size * inp_size * sizeof(int));
    hipMemcpy(Dstreetmap, streetmap, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&Dscaled_map, MAP_SIZE * MAP_SIZE * sizeof(map));
    hipMalloc((void **)&DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(Outputmap));

    printf("loaded input  map , inp size %d \n",inp_size);
}

void street::Output_map(char *outfilename)
{

    hipMemcpy(Outputmap, DOutputmap, sizeof(Outputmap), hipMemcpyDeviceToHost);

    // if a phase is ended. output current map.
    FILE *outfile = fopen(outfilename, "ab");

    fwrite(Outputmap, sizeof(int), MAP_SIZE * MAP_SIZE, outfile);

    fclose(outfile);

    return;
}