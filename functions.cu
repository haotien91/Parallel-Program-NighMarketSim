#include "hip/hip_runtime.h"
#include "functions.cuh"

__device__ void scale_map(int *Dstreetmap, map *Dscaled_map, pos position)
{
    int scale_x = position.x;
    int scale_y = position.y;
    int inp_x = scale_x / SCALE_SIZE;
    int inp_y = scale_y / SCALE_SIZE;

    if (Dstreetmap[C(inp_x, inp_y, MAP_SIZE / SCALE_SIZE)] == 1)
    {
        Dscaled_map[C(scale_x, scale_y, MAP_SIZE)].vis = EMPTY;
    }
    else
    {
        Dscaled_map[C(scale_x, scale_y, MAP_SIZE)].vis = BLOCKED;
    }

    return;
}

__device__ void output_map(map *Dscaled_map, int *DOutput_map, pos position)
{
    if (Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis >= 0) // Some people on this spot
        DOutput_map[C(position.x, position.y, MAP_SIZE)] = 1;      //

    return;
}
__global__ void set(int *Dstreetmap, map *Dscaled_map)
{
    // scale map
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));
    scale_map(Dstreetmap, Dscaled_map, position);
    person *p;
    // set people
    if (position.x == 0 && position.y < NUMOFPEOPLE)
    {
        // should set people

        int direction = LEFT;
        preference prefer(4, 4, 90, 2);
        // prefer.set_preference(4,4,90,2);

        p = new person(direction, position, 1, prefer);
        Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis = direction;
        Dscaled_map[C(position.x, position.y, MAP_SIZE)].buffer[direction] = p;
    }
    return;
}

__global__ void decide(map *Dscaled_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));

    // printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);

    if (Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis > -1)
    {
        // have person
        Dscaled_map[C(position.x, position.y, MAP_SIZE)].buffer[Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis]->decide(Dscaled_map);
    }
    return;
}

__global__ void run(map *Dscaled_map)
{

    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));

    // walk
    if (Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis > -1)
    {
        // have person
        Dscaled_map[C(position.x, position.y, MAP_SIZE)].buffer[Dscaled_map[C(position.x, position.y, MAP_SIZE)].vis]->walk(Dscaled_map);
    }
    return;
}

__global__ void check(map *Dscaled_map, int *DOutput_map)
{

    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));
    map location = Dscaled_map[C(position.x, position.y, MAP_SIZE)];

    // walk
    if (location.vis > -1)
    {
        int tmp[4];
        int counter = 0;
        for (int i = 0; i < 4; i++)
        {
            if (location.buffer[i] != NULL)
            {
                tmp[counter++] = i;
            }
        }

        hiprandState state;
        hiprand_init(clock64(), counter, 0, &state);

        int random_pos = hiprand(&state) % counter;
        int random_val = tmp[random_pos];

        for (int i = 0; i < 4; i++)
        {
            if (location.buffer[i] != NULL)
            {
                if (i != random_val)
                {
                    // go back to previous_position
                    location.buffer[i]->walk_back(Dscaled_map);
                    location.buffer[i]->next_position = location.buffer[i]->position;

                    // set to null
                    location.buffer[i] = NULL;
                }
                else
                {
                    location.buffer[i]->position = location.buffer[i]->next_position;
                }
            }
        }
        location.vis = random_val;
        Dscaled_map[C(position.x, position.y, MAP_SIZE)] = location;
    }

    // Delete people if people is out
    //  code here

    // create people at a rate , if this is a startpoint , (之後再說)

    output_map(Dscaled_map, DOutput_map, position);
}

// testing

__global__ void test_write(int* Dstreetmap, map *Dscaled_map, int *DOutput_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));
    output_map(Dscaled_map, DOutput_map, position);

    if (position.x == 0 && position.y == 0)
    {
        printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);
        printf("This is Dstreetmap\n");
        for (int i = 0; i < 64; i++)
        {
            for (int j = 0; j < 64; j++)
            {
                printf("%d ", Dstreetmap[C(i, j, MAP_SIZE)]);
            }
            // printf("64\n");
            printf("\n");
        }
        printf("greets from Dstreetmap\n");
    }

}

__global__ void test_out(int *DOutput_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));

    // printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);

    // check
    if (position.x == 0 && position.y == 0)
    {
        printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);
        printf("This is DOutput_map\n");
        for (int i = 0; i < 64; i++)
        {
            for (int j = 0; j < 64; j++)
            {
                printf("%d ", DOutput_map[C(i, j, MAP_SIZE)]);
            }
            // printf("64\n");
            printf("\n");
        }
        printf("greets from Doutput_map\n");
    }
}
