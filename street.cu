#include "street.cuh"

void street::Load_map(char *infilename)
{
    FILE *file = fopen(infilename, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    streetmap = (int *)malloc(inp_size * inp_size);
    Outputmap = (int *)malloc(MAP_SIZE * MAP_SIZE);

    for (int i = 0; i < inp_size; i++)
    {
        fread(&streetmap, sizeof(int), inp_size * inp_size, file);
    }

    // throw to gpu

    hipMalloc((void **)&Dstreetmap, inp_size * inp_size * sizeof(int));
    hipMemcpy(Dstreetmap, streetmap, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&Dscaled_map, MAP_SIZE * MAP_SIZE * sizeof(map));
    hipMalloc((void **)&DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(Outputmap));
}

void street::Output_map(char *outfilename)
{

    hipMemcpy(Outputmap, DOutputmap, sizeof(Outputmap), hipMemcpyDeviceToHost);

    // if a phase is ended. output current map.
    FILE *outfile = fopen(outfilename, "ab");

    fwrite(&Outputmap, sizeof(int), MAP_SIZE * MAP_SIZE, outfile);

    fclose(outfile);

    return;
}