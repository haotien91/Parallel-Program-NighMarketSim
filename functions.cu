#include "hip/hip_runtime.h"
#include"functions.cuh"


__device__ void scale_map(int * Dstreetmap,map * Dscaled_map,pos position)
{
    int scale_x = position.x;
    int scale_y = position.y ; 
    int inp_x = scale_x/SCALE_SIZE;
    int inp_y = scale_y/SCALE_SIZE;

    if(Dstreetmap[ C(inp_x,inp_y,MAP_SIZE/SCALE_SIZE)]  == 1)
    {
        Dscaled_map[C(scale_x,scale_y,MAP_SIZE)].vis = -1;
    }
    else
    {
        Dscaled_map[C(scale_x,scale_y,MAP_SIZE)].vis = -2;

    }

    return;
}

__device__ void output_map(map * Dscaled_map,int* DOutput_map,pos position)
{
   if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1)DOutput_map[C(position.x,position.y,MAP_SIZE)] = 1;

    return ;
}
__global__ void set(int * Dstreetmap,map * Dscaled_map)
{
    // scale map 
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    
    scale_map(Dstreetmap,Dscaled_map,position);

   
    person * p ;
    // set people
    if(position.x == 1 && position.y < NUMOFPEOPLE)
    {
         // should set people 

        int direction = RIGHT;
        preference prefer(4,4,45,47);
       // prefer.set_preference(4,4,90,2);

        p = new person(direction,position,1,prefer);
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis = direction;
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[direction] = p ;  
    }


    return ;
}


__global__ void decide(map * Dscaled_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );

    if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1)
    {
            // have person 
            Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis]->decide(Dscaled_map);

          //  Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis =  Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis]->direction;

    }

    
    return ;

}


__global__ void run(map * Dscaled_map)
{
  
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    
    // walk 
    if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1 )
    {
        // have person 
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis]->walk(Dscaled_map);
    }
    return ;
}


__global__ void check(map * Dscaled_map,int * DOutput_map)
{

    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    map location = Dscaled_map[C(position.x,position.y,MAP_SIZE)];

    // walk 
    if(location.vis > -1)
    {
        int tmp[4] ;
        int counter =0 ;
        for(int i = 0 ; i < 4 ; i++)
        {
            if(location.buffer[i] != NULL)
            {
                // is a moved person
                if(location.buffer[i]->next_position.x == position.x && location.buffer[i]->next_position.y == position.y)tmp[counter++] = i ;

                // person original place is gone
                else 
                {
                    location.buffer[i] = NULL;
                    location.vis = -1 ; 
                }
            }
        }
        if(counter > 0)
        {
            hiprandState state;
            hiprand_init(clock64(), 0, 0, &state);
            float myrandf = hiprand_uniform(&state);
            myrandf *= (counter);
            myrandf += UP;
            int random_pos = (int)truncf(myrandf);; //NEED TO CHANGE 
            int random_val = tmp[random_pos];

            //synchronize next position and position
            for(int i = 0 ; i < 4 ; i++)
            {
                if(location.buffer[i] != NULL)
                {
                    if(i  != random_val)
                    {
                        // go back to previous_position
                    location.buffer[i]->walk_back(Dscaled_map);
                    location.buffer[i]->next_position = location.buffer[i]->position;

                    //set to null 
                    location.buffer[i] = NULL;

                    }
                    else
                    {
                        location.buffer[i]->position = location.buffer[i]->next_position;
                        location.vis = i;
                    }
                }
            }
        }
        Dscaled_map[C(position.x,position.y,MAP_SIZE)] = location;
        
    }

    //Delete people if people is out
    // code here 

    // create people at a rate , if this is a startpoint , (之後再說)


    output_map( Dscaled_map,DOutput_map,position);
}


__global__ void test_write(int* Dstreetmap, map *Dscaled_map, int *DOutput_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));
   // output_map(Dscaled_map, DOutput_map, position);

    if (position.x == 0 && position.y == 0)
    {
        printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);
        printf("This is Dscaled_map\n");
        for (int i = 0; i < 64; i++)
        {
            for (int j = 0; j < 64; j++)
            {
                printf("%d ", Dscaled_map[C(j, i, MAP_SIZE)].vis);
            }
            // printf("64\n");
            printf("\n");
        }
        printf("greets from Dscaled_map\n");
    }

}

__global__ void test_out(int *DOutput_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x), (blockIdx.y * blockDim.y + threadIdx.y));

    // printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);

    // check
    if (position.x == 0 && position.y == 0)
    {
        printf("My blkIdx = (%d,%d), thrIdx=(%d,%d) with pos.x=%d (actual=%d), pos.y=%d (actual=%d)\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, position.x, blockIdx.x * blockDim.x + threadIdx.x, position.y, blockIdx.y * blockDim.y + threadIdx.y);
        printf("This is DOutput_map\n");
        for (int i = 0; i < 64; i++)
        {
            for (int j = 0; j < 64; j++)
            {
                printf("%d ", DOutput_map[C(i, j, MAP_SIZE)]);
            }
            // printf("64\n");
            printf("\n");
        }
        printf("greets from Doutput_map\n");
    }
}