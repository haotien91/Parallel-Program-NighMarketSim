#include "street.cuh"

void street::Load_map(char *infilename)
{
    FILE *file = fopen(infilename, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    int *tmp = (int *)malloc(inp_size * inp_size);

    for (int i = 0; i < inp_size; i++)
    {
        fread(&tmp, sizeof(int), inp_size * inp_size, file);
    }

    // throw to gpu

    hipMalloc((void **)&Dstreetmap, inp_size * inp_size * sizeof(int));
    hipMemcpy(Dstreetmap, tmp, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);
}

void street::Output_map(char *outfilename)
{
    // if a phase is ended. output current map.

    FILE *outfile = fopen(outfilename, "ab");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    for (int i = 0; i < inp_size; i++)
    {
    }

    fclose(outfile);

    return;
}