#include "hip/hip_runtime.h"
#include"functions.cuh"


__device__ void scale_map(int * Dstreetmap,map * Dscaled_map,pos position)
{
    int scale_x = position.x;
    int scale_y = position.y ; 
    int inp_x = scale_x/SCALE_SIZE;
    int inp_y = scale_y/SCALE_SIZE;

    if(Dstreetmap[ C(inp_x,inp_y,MAP_SIZE/SCALE_SIZE)]  == 1)
    {
        Dscaled_map[C(scale_x,scale_y,MAP_SIZE)].vis = -1;
    }
    else
    {
        Dscaled_map[C(scale_x,scale_y,MAP_SIZE)].vis = -2;

    }

    return;
}

__device__ void output_map(map * Dscaled_map,int* DOutput_map,pos position)
{
   if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1)DOutput_map[C(position.x,position.y,MAP_SIZE)] = 1;

    return ;
}
__global__ void set(int * Dstreetmap,map * Dscaled_map)
{
    // scale map 
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    scale_map(Dstreetmap,Dscaled_map,position);
    person * p ;
    // set people
    if(position.x == 0 && position.y < NUMOFPEOPLE)
    {
         // should set people 

        int direction = LEFT;
        preference prefer(4,4,90,2);
        p = new person(direction,position,1,prefer);
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis = direction;
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[direction] = p ;  
    }


    return ;
}

void
__global__ decide(map * Dscaled_map)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );

    if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1)
    {
            // have person 
            Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis]->decide(Dscaled_map);
    }
    return ;

}

void
__global__  run(map * Dscaled_map)
{
  
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    
    // walk 
    if(Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis > -1 )
    {
        // have person 
        Dscaled_map[C(position.x,position.y,MAP_SIZE)].buffer[Dscaled_map[C(position.x,position.y,MAP_SIZE)].vis]->walk(Dscaled_map);
    }
    return ;
}

void
__global__ check(map * Dscaled_map,int * DOutput_map)
{
    std::srand(std::time(0)); 

    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    map location = Dscaled_map[C(position.x,position.y,MAP_SIZE)];

    // walk 
    if(location.vis > -1)
    {
        std::vector<int> tmp ;
        for(int i = 0 ; i < 4 ; i++)
        {
            if(location.buffer[i] != NULL)
            {
                tmp.push_back(i);
            }
        }
        int random_pos = std::rand() % tmp.size(); 
        int random_val = tmp[random_pos];

        for(int i = 0 ; i < 4 ; i++)
        {
            if(location.buffer[i] != NULL)
            {
                if(i  != random_val)
                {
                    // go back to previous_position
                location.buffer[i]->walk_back(Dscaled_map);
                location.buffer[i]->next_position = location.buffer[i]->position;

                //set to null 
                location.buffer[i] = NULL;

                }
                else
                {
                    location.buffer[i]->position = location.buffer[i]->next_position;
                }
            }
        }
        location.vis = random_val;
        Dscaled_map[C(position.x,position.y,MAP_SIZE)] = location;
    }
    output_map( Dscaled_map,DOutput_map,position);
}
