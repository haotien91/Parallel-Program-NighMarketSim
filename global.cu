#include "hip/hip_runtime.h"
#include "global.cuh"

__device__ int preference::choose()
{
    // Define weighted probabilities for each direction
    int weights[] = {this->up, this->down, this->left, this->right}; // Adjust these values for different weights

    // Calculate total weight
    int totalWeight = 0;
    for (int weight : weights)
    {
        totalWeight += weight;
    }

    // Generate a random number between 1 and total weight
    //  std::random_device rd;
    //   std::mt19937 gen(rd());
    //   std::uniform_int_distribution<> dis(1, totalWeight);

    // assume have already set up hiprand and generated state for each thread...
    // assume ranges vary by thread index
    hiprandState state;
    hiprand_init(clock64(), 0, 0, &state);
    float myrandf = hiprand_uniform(&state);
    myrandf *= (100);
    myrandf += UP;

    int randomNum = (int)truncf(myrandf);
    ; // NEED TO CHANGE

    // Choose a direction based on weighted probabilities
    if (randomNum <= weights[0])
    {
        return UP; // Up
    }
    else if (randomNum <= weights[0] + weights[1])
    {
        return DOWN; // Down
    }
    else if (randomNum <= weights[0] + weights[1] + weights[2])
    {
        return LEFT; // Left
    }
    else
    {
        return RIGHT; // Right
    }
}
