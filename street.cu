#include "street.cuh"

int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}

void street::Load_map(char *infilename)
{
    FILE *file = fopen(infilename, "rb");
    int inp_size = MAP_SIZE / SCALE_SIZE;

    this->streetmap = (int *)malloc(inp_size * inp_size *sizeof(int));
    this->Outputmap = (int *)malloc(MAP_SIZE * MAP_SIZE *sizeof(int));

    fread(this->streetmap, sizeof(int), inp_size * inp_size, file);

    for (int i = 0; i < inp_size; i++)
    {
        for (int j = 0; j < inp_size; j++)
        {
            printf("%d ", this->streetmap[C(j, i, inp_size)]);
        }
        printf("\n");
    }

    // throw to gpu

    hipMalloc((void **)&this->Dstreetmap, inp_size * inp_size * sizeof(int));
    hipMemcpy(this->Dstreetmap, this->streetmap, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&this->Dscaled_map, MAP_SIZE * MAP_SIZE * sizeof(map));
    hipMalloc((void **)&this->DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(int));

    printf("loaded input  map , inp size %d \n", inp_size);
}

void street::Set_bounds()
{
    int x_count = 0 ,y_count = 0,state  ;
    int inp_size = MAP_SIZE / SCALE_SIZE;

    this->x_bounds = (int*)malloc(inp_size * inp_size * sizeof(int));
    this->y_bounds = (int*)malloc(inp_size * inp_size * sizeof(int));
    memset(this->x_bounds,-1, inp_size * inp_size * sizeof(int));
    memset(this->y_bounds,-1, inp_size * inp_size * sizeof(int));
    
    for(int i = 0 ; i < inp_size ; i++ )
    {
        state = this->streetmap[C(0,i,inp_size)];
        x_count = 0;
        for(int j = 0 ; j < inp_size ;j++)
        {
            if(this->streetmap[C(j,i,inp_size)] != state)
            {
                if(state == 0)
                {
                   if(this->x_bounds[C(x_count,i,inp_size)] != j-1)this->x_bounds[C(x_count++,i,inp_size)]  = j-1;
                   state  = 1;
                }
                else
                {
                    this->x_bounds[C(x_count++,i,inp_size)] = j;
                    state  = 0;
                }
            }
        }
        qsort(this->x_bounds,x_count,sizeof(int),cmpfunc);
    }

    for(int i = 0 ; i < inp_size ; i++ )
    {
        state = this->streetmap[C(i,0,inp_size)];
        y_count = 0;
        for(int j = 0 ; j < inp_size ;j++)
        {
            if(this->streetmap[C(i,j,inp_size)] != state)
            {
                if(state == 0)
                {
                   if(this->y_bounds[C(y_count,i,inp_size)] != j-1)this->y_bounds[C(y_count++,i,inp_size)] = j-1;
                   state  = 1;
                }
                else
                {
                    this->y_bounds[C(y_count++,i,inp_size)] = j;
                    state  = 0;
                }
            }
        }
        qsort(this->y_bounds,y_count,sizeof(int),cmpfunc);
    }

    printf("y_bounds:\n");
    for(int i= 0 ; i< inp_size ;i++)
    {
        for(int j =0 ; j< inp_size ;j++)
        {
            printf("%d ", this->y_bounds[C(j,i,inp_size)]);
        }
        printf("\n");
    }

    printf("x_bounds:\n"); 
    for(int i= 0 ; i< inp_size ;i++)
    {
        for(int j =0 ; j< inp_size ;j++)
        {
            printf("%d ", this->x_bounds[C(j,i,inp_size)]);
        }
        printf("\n");
    }

    //SCALING 
    for(int i = 0 ; i < inp_size * inp_size ; i++ )
    {
        this -> x_bounds[i] = this -> x_bounds[i] * SCALE_SIZE + (SCALE_SIZE-1);
        this -> y_bounds[i] = this -> y_bounds[i] * SCALE_SIZE + (SCALE_SIZE-1);
    }
    
    hipMalloc((void **)&this->Dx_bounds, inp_size * inp_size * sizeof(int));
    hipMemcpy(this->Dx_bounds, this->x_bounds, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&this->Dy_bounds, inp_size * inp_size * sizeof(int));
    hipMemcpy(this->Dy_bounds, this->y_bounds, inp_size * inp_size * sizeof(int), hipMemcpyHostToDevice);

    


}
void street::Output_map(char *outfilename)
{

    hipMemcpy(this->Outputmap, this->DOutputmap, MAP_SIZE * MAP_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // char newline = '\n';

    // if a phase is ended. output current map.
    FILE *outfile = fopen(outfilename, "ab");

    fwrite(this->Outputmap, sizeof(int), MAP_SIZE * MAP_SIZE, outfile);
    // fwrite(&newline, sizeof(char), 1, outfile);

    fclose(outfile);

    return;
}

void street::Output_size(char *outfilename)
{
    FILE *outfile = fopen(outfilename, "w");

    // char blank = ' ';
    // char newline = '\n';

    fwrite(&this->size, sizeof(int), 1, outfile);
    // fwrite(&blank, sizeof(char), 1, outfile);
    // fwrite(&this->height, sizeof(int), 1, outfile);
    // fwrite(&newline, sizeof(char), 1, outfile);
}
