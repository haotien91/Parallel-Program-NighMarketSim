#include "hip/hip_runtime.h"
#include "global.cuh"

__device__ void preference::set_weights(int * Dx_bounds, int * Dy_bounds,pos position)
{
    // // find left, right bound 
    int left,right,up,down;
    for(int i = 0 ; i < MAP_SIZE/SCALE_SIZE ; i++)
    {
        if (position.x <= Dx_bounds[position.y][i] && Dx_bounds[position.y][i] != -1)
        {
            right = Dx_bounds[position.y][i];
            if(i ==0)left = -100000000;
            else left = Dx_bounds[position.y][i-1];


            break;
        }
        if(Dx_bounds[position.y][i] == -1 )
        {
            right = 100000000;
            if(i ==0)left = -100000000;
            else left = Dx_bounds[position.y][i-1];
            //set to map bound
            break;
        }
    }
    

    // // find left, right bound 
    for(int i = 0 ; i < MAP_SIZE/SCALE_SIZE ; i++)
    {
        if (position.y <= Dy_bounds[position.x][i] && Dx_bounds[position.x][i] != -1)
        {
            down = Dx_bounds[position.x][i];
            if(i ==0)up = -100000000;
            else up = Dx_bounds[position.y][i-1];

            break;
        }
        if(Dy_bounds[position.x][i] == -1 )
        {
            down = 100000000;
            if(i ==0)up = -100000000;
            else up = Dx_bounds[position.y][i-1];
            //set to map bound
            break;
        }
    }
    int vecs = {position.y - up,down - position.y,position.x - left,right - position.x};
   
    vecs[BACK] = 0 ;

    // get max vecs 
    int  max_val = 0 ,max_ind;
    for(int i = 0 ; i< 4; i++)
    {
        if(max_val <= vecs[i])
        {
            max_val =vecs[i];
            max_ind = i;
        }
    }
    
    float turn_prob = 0 ;
    // if diff , get turn prob 
    if(max_ind != this->heading)
    {

        turn_prob = 10 + ( 10 > float(vecs[heading]) ? (10 - float(vecs[heading])) : 0  );
    }
    hiprandState state;
    hiprand_init(clock64(), C(position.x,position.y,MAP_SIZE) , 0, &state);
    float myrandf = hiprand_uniform(&state);

    myrandf *= (100);
    if(my_randf < turn_prob)
    {
        //turn 
        return max_ind;
    }
    else return heading;
}
__device__ int preference::choose(int * Dx_bounds, int * Dy_bounds)
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    // Define weighted probabilities for each direction

    int new_heading = set_weights(Dx_bounds,Dy_bounds,position);

    
    
    weights = this->heading_weights[new_heading];

    

    // Calculate total weight
    int totalWeight = 0;
    for (int i = 0 ; i< 4; i++)
    {
        totalWeight += weights[i];
    }

    // Generate a random number between 1 and total weight
    //  std::random_device rd;
    //   std::mt19937 gen(rd());
    //   std::uniform_int_distribution<> dis(1, totalWeight);

    // assume have already set up hiprand and generated state for each thread...
    // assume ranges vary by thread index
    hiprandState state;
    hiprand_init(clock64(), C(position.x,position.y,MAP_SIZE) , 0, &state);
    float myrandf = hiprand_uniform(&state);
    myrandf *= (100);
    myrandf += UP;

    int randomNum = (int)truncf(myrandf);
    ; // NEED TO CHANGE

    // Choose a direction based on weighted probabilities
    if (randomNum <= weights[0])
    {
        return UP; // Up
    }
    else if (randomNum <= weights[0] + weights[1])
    {
        return DOWN; // Down
    }
    else if (randomNum <= weights[0] + weights[1] + weights[2])
    {
        return LEFT; // Left
    }
    else
    {
        return RIGHT; // Right
    }
}
