#include "hip/hip_runtime.h"
#include "global.cuh"


__device__ void preference::set_weight(int vis_val)
{
  
    int new_heading  = vis_val - 4 ;  

    this->heading = new_heading;
    return ;
}
__device__ int preference::choose()
{
    pos position((blockIdx.x * blockDim.x + threadIdx.x) , (blockIdx.y * blockDim.y + threadIdx.y) );
    // Define weighted probabilities for each direction
    int * weights = this->heading_weights[this->heading]; // Adjust these values for different weights

    // Calculate total weight
    int totalWeight = 0;
    for (int i = 0 ; i< 4; i++)
    {
        totalWeight += weights[i];
    }

    // Generate a random number between 1 and total weight
    //  std::random_device rd;
    //   std::mt19937 gen(rd());
    //   std::uniform_int_distribution<> dis(1, totalWeight);

    // assume have already set up hiprand and generated state for each thread...
    // assume ranges vary by thread index
    hiprandState state;
    hiprand_init(clock64(), C(position.x,position.y,MAP_SIZE) , 0, &state);
    float myrandf = hiprand_uniform(&state);
    myrandf *= (100);
    myrandf += UP;

    int randomNum = (int)truncf(myrandf);
    ; // NEED TO CHANGE

    // Choose a direction based on weighted probabilities
    if (randomNum < weights[0])
    {
        return UP; // Up
    }
    else if (randomNum < weights[0] + weights[1])
    {
        return DOWN; // Down
    }
    else if (randomNum < weights[0] + weights[1] + weights[2])
    {
        return LEFT; // Left
    }
    else
    {
        return RIGHT; // Right
    }
}
